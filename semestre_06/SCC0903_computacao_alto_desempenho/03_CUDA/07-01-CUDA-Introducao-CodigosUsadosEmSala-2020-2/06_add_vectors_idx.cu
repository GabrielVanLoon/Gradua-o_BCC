
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define N   10000
#define TPB 128 /* indica o no. de threads por bloco */

__global__ void add( int *a, int *b, int *c ) {
    /*
     * threadIdx.x contém o Id da thread a ser executada
     * blockIdx.x contém o Id do bloco
     * blockDim.x contém o nr de threads utilizadas em cada bloco
     * gridDim.x contém o nr de blocos utilizados em um grid
     */
	 
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    // this thread handles the data at its thread id

	if(tid<N){
	        c[tid] = a[tid] + b[tid];
	}
    /**
     * O While adapta a funcao para percorrer vetor maior do que o alocado
     */

//    while (tid < N){

//        c[tid] = a[tid] + b[tid];
//    	tid+= blockDim.x * gridDim.x;
//    }
}

int main( void ) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int));
    hipMalloc( (void**)&dev_b, N * sizeof(int));
    hipMalloc( (void**)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy( dev_b, b, N * sizeof(int),hipMemcpyHostToDevice);

    /**
     *  Aloca uma quantidade maior de blocos para o processamento dos dados
     *  Função add não utiliza dados desnecessários
     */
	printf("N=%d, TPB=%d, Nr Blocos=%d \n", N, TPB, ((N + TPB - 1)/ TPB));
	
    add<<<((N+TPB-1)/TPB),TPB>>>(dev_a,dev_b,dev_c);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost);

    // display the results
    for (int i=0; i<N; i++) {
	if(i%1000==0)
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

	getchar();

    return 0;
}
